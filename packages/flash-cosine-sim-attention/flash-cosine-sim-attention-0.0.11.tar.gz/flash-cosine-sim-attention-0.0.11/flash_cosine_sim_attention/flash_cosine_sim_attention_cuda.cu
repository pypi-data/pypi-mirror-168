#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <c10/cuda/CUDAGuard.h>

#include <torch/extension.h>

// error handler
// from https://leimao.github.io/blog/Proper-CUDA-Error-Checking

#define CHECK_LAST_CUDA_ERROR() check(__FILE__, __LINE__)
void check(const char* file, const int line)
{
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        std::cerr << "CUDA Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}

#define ACCESSOR(x, n, type) x.packed_accessor32<type, n, torch::RestrictPtrTraits>()

// type alias

template <typename scalar_t, int dims>
using PackedAccessor = torch::PackedTensorAccessor32<scalar_t, dims, torch::RestrictPtrTraits>;

// helper functions

__host__ __device__ int cdiv(int numer, int denom) {
    return (numer + denom - 1) / denom;
}

__host__ __device__ int next_multiple_of(int num, int multiple_of) {
    return cdiv(num, multiple_of) * multiple_of;
}

__host__ __device__ int next_pow_2(int n) {
    int i = 1;
    while(i < n)
        i *= 2;
    return i;
}

bool divisible_by(int num, int denom) {
    return (num % denom) == 0;
}

// constants

__constant__ float NULL_FLOAT_VALUE = -3.14159e5;

// shared memory fragment

template<typename T>
struct smem_fragment {
    T* smem;
    int N;
    int M;
    bool transposed = false;

    __device__ smem_fragment(char* shared_base, int N, int M)
      : smem(reinterpret_cast<T*>(shared_base)), N(N), M(M) { }

    __device__ void load(const T* gmem) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            smem[i] = gmem[i];
        }
    }

    __device__ T get(int index) {
        return smem[index];
    }

    __device__ int get_row() {
        return transposed ? N : M;
    }

    __device__ int get_col() {
        return transposed ? M : N;
    }

    __device__ T get_transpose(int index) {
        int i = index % N;
        int j = index / M;
        return smem[i * M + j];
    }

    __device__ void transpose_with(smem_fragment smem_fragment_buffer) {
        T* buffer = smem_fragment_buffer.smem;

        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int y = i / get_row();
            int x = i % get_row();
            buffer[x * get_col() + y] = smem[i];
        }

        __syncthreads();

        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            smem[i] = buffer[i];
        }

        __syncthreads();

        transposed = !transposed;
    }

    template<typename accessor>
    __device__ void load(accessor gmem, int tile_y, int max_y) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int x = i % M;
            int y = i / M;
            int gmem_y = y + tile_y * N;

            if (gmem_y >= max_y)
                continue;

            smem[i] = gmem[gmem_y][x];
        }
    }

    template<typename accessor>
    __device__ void load_transpose(accessor gmem, int tile_y, int max_y) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int y = i % M;
            int x = i / M;
            int gmem_y = x + tile_y * N;

            if (gmem_y >= max_y)
                continue;

            smem[y * N + x] = gmem[gmem_y][y];
        }

        transposed = true;
    }

    template<typename accessor, typename accessor_mask>
    __device__ void load_transpose(accessor gmem, int tile_y, bool has_mask, accessor_mask mask, int max_y) {
        if (!has_mask)
            return load_transpose(gmem, tile_y, max_y);

        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int y = i % M;
            int x = i / M;
            int gmem_y = x + tile_y * N;

            if (y == 0 && !mask[gmem_y]) {
                smem[y * N + x] = NULL_FLOAT_VALUE;
                continue;
            }

            if (gmem_y >= max_y)
                continue;

            smem[y * N + x] = gmem[gmem_y][y];
        }

        transposed = true;
    }

    template<typename accessor>
    __device__ void store(accessor gmem, int tile_y, int max_y) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int x = i % M;
            int y = i / M;
            int gmem_y = y + tile_y * N;

            if (gmem_y >= max_y) {
                continue;
            }

            gmem[gmem_y][x] = smem[i];
        }
    }

    __device__ unsigned size() {
        return N * M;
    }

    __device__ char* next() {
        return reinterpret_cast<char*>(smem + size());
    }
};

// mma

template<typename scalar_t, int tmpl_N_thread, int tmpl_M_thread>
struct mma_warp_tile {
    static constexpr int N_thread = tmpl_N_thread;
    static constexpr int M_thread = tmpl_M_thread;

    // Thread layout within a warp:
    static constexpr int N_warp = 8;
    static constexpr int M_warp = 4;
    static_assert(N_warp * M_warp == 32);

    // Warp layout within a block:
    static constexpr int N_block = 2;
    static constexpr int M_block = 4;
    static_assert(N_block * M_block * N_warp * M_warp == 256); // blockDim.x

    // Dimensions of the tile, in threads:
    static constexpr int N_tile = N_warp * N_block * N_thread;
    static constexpr int M_tile = M_warp * M_block * M_thread;

    static constexpr float MASK_VALUE = -1e8;

    // Registers:
    float A_frag[N_thread];            // N x 1 fragment
    float B_frag[M_thread];            // 1 x M fragment
    float C_frag[N_thread * M_thread]; // N x M fragment


    int warp_x;   // x offset of the warp within the block tile
    int warp_y;   // y offset of the warp within the block tile
    int thread_x; // x offset of the thread within the warp tile
    int thread_y; // y offset of the thread within the warp tile

    __device__ mma_warp_tile() {
        int warp_id = threadIdx.x / 32;
        warp_x = (warp_id % M_block);
        warp_y = (warp_id / M_block);

        int lane_id = threadIdx.x % 32;
        thread_x = warp_x * M_warp * M_thread + lane_id % M_warp;
        thread_y = warp_y * N_warp * N_thread + lane_id / M_warp;
    }

    // Initialize C to all zeros

    __device__ void zero() {
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = 0.f;
        }
    }

    // Performs C = A * B + C

    __device__ void mma_full(
        smem_fragment<scalar_t> A_sm,
        smem_fragment<scalar_t> B_sm,
        int k,
        bool has_mask,
        bool transpose_a,
        bool transpose_b
    ) {
        // Load a N x 1 fragment of A from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            int sm_idx = i * N_warp + thread_y + k * A_sm.get_row();
            A_frag[i] = !transpose_a ? A_sm.get(sm_idx) : A_sm.get_transpose(sm_idx);
        }

        // Load a 1 x M fragment of B from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < M_thread; i++) {
            int sm_idx = i * M_warp + thread_x + k * M_tile;
            B_frag[i] = !transpose_b ? B_sm.get(sm_idx) : B_sm.get_transpose(sm_idx);
        }

        // Compute:
        #pragma unroll
        for (int j = 0; j < M_thread ; j++) {

            bool is_masked_out = false;

            if (has_mask) {
                int sm_idx = j * M_warp + thread_x;
                scalar_t maybe_mask_val = !transpose_b ? B_sm.get(sm_idx) : B_sm.get_transpose(sm_idx);
                is_masked_out = (maybe_mask_val == NULL_FLOAT_VALUE);
            }

            #pragma unroll
            for (int i = 0; i < N_thread; i++) {
                if (is_masked_out) {
                    C_frag[i * M_thread + j] = MASK_VALUE;
                } else {
                    C_frag[i * M_thread + j] += A_frag[i] * B_frag[j];
                }
            }
        }
    }

    __device__ void mma(
        smem_fragment<scalar_t> A_sm,
        smem_fragment<scalar_t> B_sm,
        int k,
        bool has_mask
    ) {
        return mma_full(A_sm, B_sm, k, has_mask, false, false);
    }

    __device__ void mma(
        smem_fragment<scalar_t> A_sm,
        smem_fragment<scalar_t> B_sm,
        int k
    ) {
        return mma_full(A_sm, B_sm, k, false, false, false);
    }

    __device__ void mma_transpose_a(
        smem_fragment<scalar_t> A_sm,
        smem_fragment<scalar_t> B_sm,
        int k
    ) {
        return mma_full(A_sm, B_sm, k, false, true, false);
    }

    // Perform a pointwise operation, specified by the given lambda, on C

    template<typename F>
    __device__ void pointwise(F&& op) {
        #pragma unroll
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = op(C_frag[i]);
        }
    }

    template<typename F>
    __device__ void pointwise(int tile_y, int tile_x, F&& op) {
        #pragma unroll
        for (int i = 0; i < N_thread * M_thread; i++) {
            int global_row = tile_y * N_tile + (i / M_thread) * N_warp + thread_y;
            int global_col = tile_x * M_tile + (i % M_thread) * M_warp + thread_x;

            C_frag[i] = op(C_frag[i], global_row, global_col);
        }
    }

    // copy from shared memory to registers in C

    __device__ void load(scalar_t* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_frag[i * M_thread + j]
                    = C_sm_ptr[(thread_y + i * N_warp) * M_tile + j * M_warp + thread_x];
            }
        }
    }

    // copy from registers to shared memory

    __device__ void store(scalar_t* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[(thread_y + i * N_warp) * M_tile + j * M_warp + thread_x]
                    = C_frag[i * M_thread + j];
            }
        }
    }

    __device__ void add_to(scalar_t* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[(thread_y + i * N_warp) * M_tile + j * M_warp + thread_x]
                    += C_frag[i * M_thread + j];
            }
        }
    }

    __device__ void store_transpose(scalar_t* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[thread_y + i * N_warp + (j * M_warp + thread_x) * N_tile]
                    = C_frag[i * M_thread + j];
            }
        }
    }

    template<typename accessor>
    __device__ void store(accessor gmem, int tile_x_offset, int tile_y_offset, int max_x, int max_y) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                int gmem_y = thread_y + i * N_warp + tile_y_offset;
                int gmem_x = thread_x + j * M_warp + tile_x_offset;

                if (gmem_y >= max_y || gmem_x >= max_x)
                    continue;

                gmem[gmem_y][gmem_x] = C_frag[i * M_thread + j];
            }
        }
    }

    // atomic add from registers go global memory

    template<typename accessor>
    __device__ void atomic_add(accessor gmem, int tile_x_offset, int tile_y_offset, int max_x, int max_y) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                int gmem_y = thread_y + i * N_warp + tile_y_offset;
                int gmem_x = thread_x + j * M_warp + tile_x_offset;

                if (gmem_y >= max_y || gmem_x >= max_x)
                    continue;

                atomicAdd((float*) &gmem[gmem_y][gmem_x], C_frag[i * M_thread + j]);
            }
        }
    }
};

template<typename scalar_t>
struct out_mma_warp_tile {
    // How much data is processed by a single thread:
    static constexpr int N_thread = 4;
    static constexpr int M_thread = 4;

    // Thread layout within a warp:
    static constexpr int N_warp = 8;
    static constexpr int M_warp = 4;
    static_assert(N_warp * M_warp == 32);

    // Warp layout within a block:
    static constexpr int N_block = 2;
    static constexpr int M_block = 4;
    static_assert(N_block * M_block * N_warp * M_warp == 256); // blockDim.x

    // Dimensions of the tile, in threads:
    static constexpr int N_tile = N_warp * N_block * N_thread;
    static constexpr int M_tile = M_warp * M_block * M_thread;

    static constexpr float EPS = 1e-10;

    // Registers:
    float A_frag[N_thread];            // N x 1 fragment
    float B_frag[M_thread];            // 1 x M fragment
    float L_frag[N_thread];            // N x 1 fragment
    float C_frag[N_thread * M_thread]; // N x M fragment

    int warp_x;   // x offset of the warp within the block tile
    int warp_y;   // y offset of the warp within the block tile
    int thread_x; // x offset of the thread within the warp tile
    int thread_y; // y offset of the thread within the warp tile

    __device__ out_mma_warp_tile() {
        int warp_id = threadIdx.x / 32;
        warp_x = (warp_id % M_block);
        warp_y = (warp_id / M_block);

        int lane_id = threadIdx.x % 32;
        thread_x = warp_x * M_warp * M_thread + lane_id % M_warp;
        thread_y = warp_y * N_warp * N_thread + lane_id / M_warp;
    }

    // Initialize C to all zeros
    __device__ void zero() {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            L_frag[i] = 0.f;
        }

        #pragma unroll
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = 0.f;
        }
    }

    // Performs C = A * B + C
    __device__ void mma(
        smem_fragment<scalar_t> A_sm,
        smem_fragment<scalar_t> B_sm,
        int k
    ) {
        // Load a N x 1 fragment of A from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            A_frag[i] = A_sm.get(i * N_warp + thread_y + k * N_tile);
        }

        // Load a 1 x M fragment of B from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < M_thread; i++) {
            B_frag[i] = B_sm.get(i * M_warp + thread_x + k * M_tile);
        }

        // Compute:
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            L_frag[i] += A_frag[i];

            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_frag[i * M_thread + j] += A_frag[i] * B_frag[j];
            }
        }
    }

    // Copy C from registers to shared memory
    __device__ void store(scalar_t* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            float inv_rowsum = 1.f / max(L_frag[i], EPS);

            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[(thread_y + i * N_warp) * M_tile + j * M_warp + thread_x]
                  = C_frag[i * M_thread + j] * inv_rowsum;
            }
        }
    }

    template<typename accessor>
    __device__ void store(accessor gmem, int tile_y, int max_y) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            float inv_rowsum = 1.f / max(L_frag[i], EPS);

            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                int gmem_y = thread_y + i * N_warp + tile_y * N_tile;
                int gmem_x = thread_x + j * M_warp;

                if (gmem_y >= max_y)
                    continue;

                gmem[gmem_y][gmem_x] = C_frag[i * M_thread + j] * inv_rowsum;
            }
        }
    }

    template<typename accessor>
    __device__ void store_rowsum(accessor gmem, int tile_y, int max_y) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            int gmem_idx = tile_y * N_tile + i * N_warp + thread_y;

            if (gmem_idx >= max_y)
                continue;

            gmem[gmem_idx] = L_frag[i];
        }
    }

    __device__ void store_transpose(scalar_t* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            float inv_rowsum = 1.f / max(L_frag[i], EPS);

            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[thread_y + i * N_warp + (j * M_warp + thread_x) * N_tile]
                    = C_frag[i * M_thread + j] * inv_rowsum;
            }
        }
    }
};

// forward kernel

template<typename scalar_t>
__global__ void forward_kernel(
    const PackedAccessor<scalar_t, 4> Q,
    const PackedAccessor<scalar_t, 4> K,
    const PackedAccessor<scalar_t, 4> V,
          PackedAccessor<scalar_t, 4> O,
          PackedAccessor<scalar_t, 3> L,
    const PackedAccessor<bool, 2> mask,
    const PackedAccessor<scalar_t, 3> attn_bias,
    const float scale,
    const bool causal,
    const bool has_mask,
    const bool has_attn_bias,
    const bool need_store_rowsum
) {

    const int q_seq_len = Q.size(2);
    const int k_seq_len = K.size(2);
    const int qk_seq_len_diff = k_seq_len - q_seq_len;  // for calculating causality when query and key lengths differ

    const int D = Q.size(3);
    const int E = V.size(3);


    const int batch = blockIdx.y / Q.size(1);
    const int heads = blockIdx.y % Q.size(1);

    // shortcut accessor

    auto Q_ = Q[batch][heads];
    auto K_ = K[batch][heads];
    auto V_ = V[batch][heads];
    auto O_ = O[batch][heads];
    auto L_ = L[batch][heads];
    auto attn_bias_ = attn_bias[heads];

    // mma

    mma_warp_tile<scalar_t, 4, 4> QK_mma;
    out_mma_warp_tile<scalar_t> out_mma;

    // tiles

    const int num_col_tiles = cdiv(k_seq_len, QK_mma.M_tile);
    const int tile_y = blockIdx.x;

    // shared memory

    extern __shared__ char _shared_mem[];

    smem_fragment<scalar_t> Q_sm{_shared_mem, QK_mma.N_tile, D};
    smem_fragment<scalar_t> A_sm{Q_sm.next(), QK_mma.N_tile, QK_mma.M_tile};
    smem_fragment<scalar_t> K_sm{A_sm.next(), QK_mma.M_tile, D};
    smem_fragment<scalar_t> V_sm{A_sm.next(), QK_mma.M_tile, E};

    // helper variables

    scalar_t bias;

    // start loop

    out_mma.zero();

    Q_sm.load_transpose(Q_, tile_y, q_seq_len);

    for (int tile_x = 0; tile_x < num_col_tiles; tile_x++) {
        if (causal && (QK_mma.M_tile * tile_x - qk_seq_len_diff) >= (QK_mma.N_tile * (tile_y + 1)))
            continue;

        K_sm.load_transpose(K_, tile_x, has_mask, mask[batch], k_seq_len);

        __syncthreads();

        QK_mma.zero();

        for (int d = 0; d < D; d++) {
            QK_mma.mma(Q_sm, K_sm, d, has_mask);
        }

        QK_mma.pointwise(tile_y, tile_x, [&](scalar_t el, int global_row, int global_col) {

            if (global_row >= q_seq_len ||
                global_col >= k_seq_len ||
                causal && (global_row < (global_col - qk_seq_len_diff)))
                return 0.f;

            bias = has_attn_bias ? attn_bias_[global_row][global_col] : (scalar_t) 0.f;

            return __expf((scale * el + bias) - scale); 
        });

        QK_mma.store_transpose(A_sm.smem);

        __syncthreads();

        V_sm.load(V_, tile_x, k_seq_len);

        __syncthreads();

        for (int d = 0; d < QK_mma.M_tile; d++) {
            out_mma.mma(A_sm, V_sm, d);
        }

        __syncthreads();
    }

    out_mma.store(O_, tile_y, q_seq_len);

    if (need_store_rowsum)
        out_mma.store_rowsum(L_, tile_y, q_seq_len);
}

// forwards c++ function

std::vector<at::Tensor> flash_cosine_sim_attention_forward(
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor mask,
    torch::Tensor attn_bias,
    float scale,
    bool causal,
    bool need_store_rowsum
) {
    auto query_device = device_of(Q);
    const at::cuda::OptionalCUDAGuard device_guard(query_device);

    const int batch = Q.size(0);
    const int heads = Q.size(1);
    const int N = Q.size(2);
    const int M = K.size(2);
    const int D = Q.size(3);
    const int E = V.size(3);

    auto options = torch::TensorOptions().device(query_device).dtype(Q.scalar_type());

    auto O = at::empty({batch, heads, N, E}, options);
    auto L = at::empty({batch, heads, need_store_rowsum ? N : 0}, options);

    const dim3 threads_per_block(256);

    const int max_feature_dimension = max(D, E);

    const bool has_attn_bias = !!attn_bias.numel();
    const bool has_mask = !!mask.numel();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(Q.scalar_type(), "forward_cosine_sim_attention_backward", ([&] {

        using mma_warp_tile_klass = mma_warp_tile<scalar_t, 4, 4>;

        const dim3 blocks(cdiv(N, mma_warp_tile_klass::N_tile), batch * heads);

        const unsigned shared_mem_size = (mma_warp_tile_klass::N_tile * max_feature_dimension +
                                          mma_warp_tile_klass::M_tile * max_feature_dimension +
                                          mma_warp_tile_klass::N_tile * mma_warp_tile_klass::M_tile) * sizeof(scalar_t);

        forward_kernel<scalar_t><<<blocks, threads_per_block, shared_mem_size>>>(
            ACCESSOR(Q, 4, scalar_t),
            ACCESSOR(K, 4, scalar_t),
            ACCESSOR(V, 4, scalar_t),
            ACCESSOR(O, 4, scalar_t),
            ACCESSOR(L, 3, scalar_t),
            ACCESSOR(mask, 2, bool),
            ACCESSOR(attn_bias, 3, scalar_t),
            scale,
            causal,
            has_mask,
            has_attn_bias,
            need_store_rowsum
        );
    }));

    // handle error

    hipDeviceSynchronize();

    CHECK_LAST_CUDA_ERROR();

    return { O, L };
}

// backward kernel

// backwards preprocess

// 1. do_scaled = do / rowsum
// 2. delta = rowsum(do_scaled * o)

// done by @ptillet at https://github.com/openai/triton/blob/master/python/tutorials/06-fused-attention.py

template <typename scalar_t>
__global__ void backward_preprocess(
    const PackedAccessor<scalar_t, 4> d_out,
    const PackedAccessor<scalar_t, 4> o,
    const PackedAccessor<scalar_t, 3> l,
          PackedAccessor<scalar_t, 4> d_out_scaled,
          PackedAccessor<scalar_t, 4> delta
) {
    const int heads = o.size(1);
    const int v_dim = o.size(3);

    const int batch_idx = blockIdx.x / heads;
    const int head_idx = blockIdx.x % heads;
    const int seq_idx = blockIdx.y;
    const int dim_idx = threadIdx.x;

    const int warp_id = threadIdx.x / 32;
    const int lane_id = threadIdx.x & 31;

    const unsigned mask = __ballot_sync(0xFFFFFFFFU, dim_idx < v_dim);

    float val = 0.0f;

    extern __shared__ char _shared_mem_preprocess[];

    scalar_t* sm_delta  = reinterpret_cast<scalar_t*>(&_shared_mem_preprocess);
    scalar_t* sm_do     = reinterpret_cast<scalar_t*>(&sm_delta[cdiv(v_dim, 32)]);
    scalar_t* sm_rowsum = reinterpret_cast<scalar_t*>(&sm_do[v_dim]);

    auto do_ = d_out[batch_idx][head_idx][seq_idx];
    auto o_ = o[batch_idx][head_idx][seq_idx];
    auto l_ = l[batch_idx][head_idx];
    auto do_scaled_ = d_out_scaled[batch_idx][head_idx][seq_idx];
    auto delta_ = delta[batch_idx][head_idx][seq_idx];

    // load rowsum into shared memory

    if (dim_idx == 0)
        sm_rowsum[0] = l_[seq_idx];

    __syncthreads();

    // load do into shared memory

    if (dim_idx < v_dim)
        sm_do[dim_idx] = do_[dim_idx] / max(sm_rowsum[0], 1e-10);

    __syncthreads();

    // store do_scaled to gmem

    if (dim_idx < v_dim)
        do_scaled_[dim_idx] = sm_do[dim_idx];

    // load do_scaled * o into registers

    if (dim_idx < v_dim)
        val = sm_do[dim_idx] * o_[dim_idx];

    // warp shuffle reduce

    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(mask, val, offset);
    }

    if (lane_id == 0)
        sm_delta[warp_id] = val;

    __syncthreads();

    if (warp_id == 0) {
        // use shared memory to reduce further across warps
        if (dim_idx < (blockDim.x / 32)) {
            val = sm_delta[lane_id];
        } else{
            val = 0;
        }

        for (int offset = 16; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(mask, val, offset);
        }

        // write out reduced rowsum(do_scaled * o)

        if (dim_idx == 0) {
            delta_[0] = val;
        }

    }
}

// main backward kernel

template <typename scalar_t>
__global__ void backward_kernel(
    const PackedAccessor<scalar_t, 4> q,
    const PackedAccessor<scalar_t, 4> k,
    const PackedAccessor<scalar_t, 4> v,    
    const PackedAccessor<bool, 2> mask,
    const PackedAccessor<scalar_t, 3> attn_bias,
          PackedAccessor<scalar_t, 4> dq,
          PackedAccessor<scalar_t, 4> dk,
          PackedAccessor<scalar_t, 4> dv,
          PackedAccessor<scalar_t, 3> d_attn_bias,
    const PackedAccessor<scalar_t, 4> d_out_scaled,
    const PackedAccessor<scalar_t, 4> delta,
    const float scale,
    const bool causal,
    const bool has_mask,
    const bool has_attn_bias,
    const bool attn_bias_requires_grad
) {

    // dimensions

    const int head = q.size(1);

    const int batch_idx = blockIdx.x / head;
    const int head_idx = blockIdx.x % head;

    const int q_seq_len = q.size(2);
    const int k_seq_len = k.size(2);
    const int qk_seq_len_diff = k_seq_len - q_seq_len;

    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    // shortcut accessors

    auto q_ = q[batch_idx][head_idx];
    auto k_ = k[batch_idx][head_idx];
    auto v_ = v[batch_idx][head_idx];
    auto dq_ = dq[batch_idx][head_idx];
    auto dk_ = dk[batch_idx][head_idx];
    auto dv_ = dv[batch_idx][head_idx];
    auto ds_ = d_attn_bias[head_idx];
    auto delta_ = delta[batch_idx][head_idx];
    auto do_ = d_out_scaled[batch_idx][head_idx];
    auto mask_ = mask[batch_idx];

    // handle attention bias

    auto attn_bias_ = has_attn_bias ? attn_bias[head_idx] : attn_bias[0];

    // variables

    scalar_t bias;

    // mma

    mma_warp_tile<scalar_t, 2, 2> mma;
    mma_warp_tile<scalar_t, 2, 4> dv_mma;
    mma_warp_tile<scalar_t, 2, 4> dk_mma;
    mma_warp_tile<scalar_t, 2, 4> dq_mma;

    // tiles

    const int num_col_tiles = cdiv(k_seq_len, mma.M_tile);
    const int num_row_tiles = cdiv(q_seq_len, mma.N_tile);

    // shared memory

    extern __shared__ char _shared_mem_backward[];

    smem_fragment<scalar_t> sm_q {_shared_mem_backward, mma.N_tile, k_dim};
    smem_fragment<scalar_t> sm_attn {sm_q.next(), mma.N_tile, mma.M_tile};
    smem_fragment<scalar_t> sm_k {sm_attn.next(), mma.M_tile, k_dim};
    smem_fragment<scalar_t> sm_v {sm_k.next(), mma.M_tile, v_dim};
    smem_fragment<scalar_t> sm_do {sm_v.next(), mma.N_tile, v_dim};

    // loop over columns

    for (int tile_x = 0; tile_x < num_col_tiles; tile_x++) {

        int col_offset = tile_x * mma.M_tile;

        // load keys and values into shared memory

        sm_k.load_transpose(k_, tile_x, has_mask, mask_, k_seq_len);

        sm_v.load_transpose(v_, tile_x, k_seq_len);

        dk_mma.zero();

        dv_mma.zero();

        // loop over rows

        for (int tile_y = 0; tile_y < num_row_tiles; tile_y++) {

            int row_offset = tile_y * mma.N_tile;

            if (causal && (col_offset - qk_seq_len_diff) >= (mma.N_tile * (tile_y + 1)))
                continue;

            // load queries and scaled do into shared memories

            sm_q.load_transpose(q_, tile_y, q_seq_len);
            sm_do.load(do_, tile_y, q_seq_len);

            __syncthreads();

            // accumulate qk similarities

            mma.zero();

            for (int d = 0; d < k_dim; d++) {
                mma.mma(sm_q, sm_k, d, has_mask);
            }

            // calculate attention

            mma.pointwise(tile_y, tile_x, [&](float el, int global_row, int global_col) {

                if (global_row >= q_seq_len ||
                    global_col >= k_seq_len ||
                    causal && (global_row < (global_col - qk_seq_len_diff)))
                    return 0.f;

                bias = has_attn_bias ? (float) attn_bias_[global_row][global_col] : 0.f;

                return expf((scale * el + bias) - scale);

            });

            mma.store(sm_attn.smem);

            __syncthreads();

            // accumulate dv to global mem

            for (int d = 0; d < mma.N_tile; d++) {
                dv_mma.mma(sm_attn, sm_do, d);
            }

            __syncthreads();

            // calculate dp

            mma.zero();

            for (int d = 0; d < v_dim; d++) {
                mma.mma_transpose_a(sm_do, sm_v, d);
            }

            __syncthreads();

            // calculate dS
            // just do things manually out in the open, as the operation is not very reusable

            #pragma unroll
            for (int i = 0; i < mma.N_thread; i++) {
                int global_row = row_offset + i * mma.N_warp + mma.thread_y;

                scalar_t row_val = delta_[global_row][0];

                #pragma unroll
                for (int j = 0; j < mma.M_thread ; j++) {
                    mma.C_frag[i * mma.M_thread + j] -= row_val;
                    mma.C_frag[i * mma.M_thread + j] *= sm_attn.get((mma.thread_y + i * mma.N_warp) * mma.M_tile + j * mma.M_warp + mma.thread_x);
                }
            }

            // store to ds_ if attention bias requires gradients

            if (attn_bias_requires_grad)
                mma.atomic_add(ds_, col_offset, row_offset, k_seq_len, q_seq_len);

            // scale

            mma.pointwise([&](scalar_t el) {
                return el * scale;
            });

            mma.store(sm_attn.smem);

            sm_q.transpose_with(sm_do);

            // calculate dk

            for (int d = 0; d < mma.N_tile; d++) {
                dk_mma.mma(sm_attn, sm_q, d);
            }

            __syncthreads();

            sm_k.transpose_with(sm_do);

            // calculate dq

            sm_q.load(dq_, tile_y, q_seq_len);

            __syncthreads();

            dq_mma.zero();

            for (int d = 0; d < mma.M_tile; d++) {
                dq_mma.mma_transpose_a(sm_attn, sm_k, d);
            }

            dq_mma.add_to(sm_q.smem);

            __syncthreads();

            sm_q.store(dq_, tile_y, q_seq_len);

            __syncthreads();

            sm_k.transpose_with(sm_do);
        }

        dv_mma.store(dv_, 0, col_offset, v_dim, k_seq_len);

        dk_mma.store(dk_, 0, col_offset, k_dim, k_seq_len);
    }
}

// backwards c++ function

std::vector<torch::Tensor> flash_cosine_sim_attention_backward(
    torch::Tensor d_out,
    torch::Tensor o,
    torch::Tensor l,
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    torch::Tensor mask,
    torch::Tensor attn_bias,
    float scale,
    bool causal,
    bool attn_bias_requires_grad
) {
    auto query_device = device_of(q);

    const at::cuda::OptionalCUDAGuard device_guard(query_device);

    const int batch = q.size(0);
    const int heads = q.size(1);
    const int seq   = q.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    const bool has_attn_bias = !!attn_bias.numel();
    const bool has_mask = !!mask.numel();

    auto options = torch::TensorOptions().device(query_device).dtype(q.scalar_type());

    // setup dq, dk, dv

    auto d_out_scaled = at::empty_like(d_out, options);
    auto delta = at::empty({batch, heads, seq, 1}, options);

    auto dq = at::zeros_like(q, options);
    auto dk = at::zeros_like(k, options);
    auto dv = at::zeros_like(v, options);

    auto db = (has_attn_bias && attn_bias_requires_grad) ? at::zeros_like(attn_bias) : at::empty({attn_bias.size(0), 0, 0}, options);

    // setup backwards preprocess call

    const dim3 backwards_preprocess_threads_per_block(next_multiple_of(v_dim, 32));

    const dim3 backwards_preprocess_blocks(batch * heads, seq);

    // setup backwards call

    const dim3 backwards_threads_per_block(256);
    const dim3 backwards_blocks(batch * heads);


    AT_DISPATCH_FLOATING_TYPES_AND_HALF(q.scalar_type(), "forward_cosine_sim_attention_backward", ([&] {

        using mma_warp_tile_klass = mma_warp_tile<scalar_t, 2, 2>;

        const int N_tile = mma_warp_tile_klass::N_tile;
        const int M_tile = mma_warp_tile_klass::M_tile;

        const unsigned backwards_preprocess_shared_mem_size = (cdiv(v_dim, 32) + v_dim + 1) * sizeof(scalar_t);

        const unsigned backwards_shared_mem_size = (  (N_tile + M_tile) * k_dim +      // q, k
                                                      (N_tile + M_tile) * v_dim +      // v, do
                                                      (N_tile * M_tile) +              // attn
                                                      N_tile                           // delta
                                                    ) * sizeof(scalar_t);

        backward_preprocess<scalar_t><<<backwards_preprocess_blocks, backwards_preprocess_threads_per_block, backwards_preprocess_shared_mem_size>>>(
            ACCESSOR(d_out, 4, scalar_t),
            ACCESSOR(o, 4, scalar_t),
            ACCESSOR(l, 3, scalar_t),
            ACCESSOR(d_out_scaled, 4, scalar_t),
            ACCESSOR(delta, 4, scalar_t)
        );

        backward_kernel<scalar_t><<<backwards_blocks, backwards_threads_per_block, backwards_shared_mem_size>>>(
            ACCESSOR(q, 4, scalar_t),
            ACCESSOR(k, 4, scalar_t),
            ACCESSOR(v, 4, scalar_t),
            ACCESSOR(mask, 2, bool),
            ACCESSOR(attn_bias, 3, scalar_t),
            ACCESSOR(dq, 4, scalar_t),
            ACCESSOR(dk, 4, scalar_t),
            ACCESSOR(dv, 4, scalar_t),
            ACCESSOR(db, 3, scalar_t),
            ACCESSOR(d_out_scaled, 4, scalar_t),
            ACCESSOR(delta, 4, scalar_t),
            scale,
            causal,
            has_mask,
            has_attn_bias,
            attn_bias_requires_grad
        );
    }));

    hipDeviceSynchronize();

    // handle error

    CHECK_LAST_CUDA_ERROR();

    return {dq, dk, dv, db};
}

// bind

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &flash_cosine_sim_attention_forward, "Flash Cosine-Sim Attention Forward");
    m.def("backward", &flash_cosine_sim_attention_backward, "Flash Cosine-Sim Attention Backward");
}
